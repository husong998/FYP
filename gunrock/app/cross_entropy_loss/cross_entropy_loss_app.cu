// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file graphsum_app.cu
 *
 * @brief gcn graphsum application
 */

#include <gunrock/gunrock.h>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph definations
#include <gunrock/graphio/graphio.cuh>
#include <gunrock/app/app_base.cuh>
#include <gunrock/app/test_base.cuh>

// single-source shortest path includes
#include <gunrock/app/cross_entropy_loss/cross_entropy_loss_enactor.cuh>
#include <gunrock/app/cross_entropy_loss/cross_entropy_loss_test.cuh>

/**
 * @brief      graphsum layer of GCN
 *
 * @param      parameters  The parameters
 * @param      graph       The graph
 * @param[in]  dim         dimension of the feature vector
 * @param      in          the input to the graphsum layer
 * @param      out         output matrix
 *
 * @tparam     GraphT      type of the graph
 * @tparam     ValueT      type of the value, double by default
 *
 * @return     time elapsed to execute
 */

namespace gunrock {
namespace app {
namespace cross_entropy_loss {

hipError_t UseParameters(util::Parameters &parameters) {
  hipError_t retval = hipSuccess;
  GUARD_CU(UseParameters_app(parameters));
  GUARD_CU(UseParameters_problem(parameters));
  GUARD_CU(UseParameters_enactor(parameters));

//  GUARD_CU(parameters.Use<std::string>(
//      "in",
//      util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::REQUIRED_PARAMETER,
//      "invalid",
//      "input file name to feature matrix", __FILE__, __LINE__
//      ));

  return retval;
}

}
}
}

template <typename GraphT, typename ValueT = typename GraphT::ValueT>
double cross_entropy_loss(gunrock::util::Parameters &parameters, GraphT &graph, const int num_nodes,
    const int num_classes, ValueT *logits, int *ground_truth, ValueT *grad, ValueT &loss) {
  typedef typename GraphT::VertexT VertexT;
  typedef gunrock::app::cross_entropy_loss::Problem<GraphT> ProblemT;
  typedef gunrock::app::cross_entropy_loss::Enactor<ProblemT> EnactorT;
  gunrock::util::CpuTimer cpu_timer;
  gunrock::util::Location target = gunrock::util::DEVICE;
  double total_time = 0;
//  if (parameters.UseDefault("quiet")) parameters.Set("quiet", true);

  // Allocate problem and enactor on GPU, and initialize them
  ProblemT problem(parameters);
  EnactorT enactor;
  problem.Init(graph, num_nodes, num_classes, logits, ground_truth);
  enactor.Init(problem, target);

  problem.Reset();
  enactor.Reset();

  cpu_timer.Start();
  enactor.Enact();
  cpu_timer.Stop();

  total_time += cpu_timer.ElapsedMillis();
  problem.Extract(grad, &loss);

  enactor.Release(target);
  problem.Release(target);

  return total_time;
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
