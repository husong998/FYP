// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file graphsum_app.cu
 *
 * @brief gcn graphsum application
 */

#include <gunrock/gunrock.h>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph definations
#include <gunrock/graphio/graphio.cuh>
#include <gunrock/app/app_base.cuh>
#include <gunrock/app/test_base.cuh>

// single-source shortest path includes
#include <gunrock/app/gcn/gcn_enactor.cuh>

/**
 * @brief      graphsum layer of GCN
 *
 * @param      parameters  The parameters
 * @param      graph       The graph
 * @param[in]  dim         dimension of the feature vector
 * @param      in          the input to the graphsum layer
 * @param      out         output matrix
 *
 * @tparam     GraphT      type of the graph
 * @tparam     ValueT      type of the value, double by default
 *
 * @return     time elapsed to execute
 */

namespace gunrock {
namespace app {
namespace gcn {

hipError_t UseParameters(util::Parameters &parameters) {
  hipError_t retval = hipSuccess;
  GUARD_CU(UseParameters_app(parameters));
  GUARD_CU(UseParameters_problem(parameters));
  GUARD_CU(UseParameters_enactor(parameters));

  GUARD_CU(parameters.Use<std::string>(
      "in",
      util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::REQUIRED_PARAMETER,
      "invalid",
      "input file name to feature matrix", __FILE__, __LINE__
      ));

  GUARD_CU(parameters.Use<int>(
      "dim",
      util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::REQUIRED_PARAMETER,
      -1,
      "feature vector dimension", __FILE__, __LINE__
      ));

  GUARD_CU(parameters.Use<std::string>(
      "out",
      util::OPTIONAL_ARGUMENT | util::SINGLE_VALUE | util::REQUIRED_PARAMETER,
      "out",
      "output file name", __FILE__, __LINE__
      ));

  return retval;
}


}
}
}

template <typename GraphT, typename ValueT = typename GraphT::ValueT>
double gcn(gunrock::util::Parameters &parameters, GraphT &graph, const int dim,
                    ValueT *in, ValueT *out) {
  typedef typename GraphT::VertexT VertexT;
  typedef gunrock::app::gcn::Problem<GraphT> ProblemT;
  typedef gunrock::app::gcn::Enactor<ProblemT> EnactorT;
  gunrock::util::CpuTimer cpu_timer;
  gunrock::util::Location target = gunrock::util::DEVICE;
  double total_time = 0;
  parameters.Set("quiet", false);

  // Allocate problem and enactor on GPU, and initialize them
  ProblemT problem(parameters);
  EnactorT enactor;
  problem.Init(graph);
  enactor.Init(problem, target);

  problem.Reset();
  enactor.Reset();

  cpu_timer.Start();
  enactor.Enact();
  cpu_timer.Stop();

  total_time += cpu_timer.ElapsedMillis();
//  problem.Extract(out);

  enactor.Release(target);
  problem.Release(target);

  return total_time;
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
