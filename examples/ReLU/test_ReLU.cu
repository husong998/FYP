// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_pr.cu
 *
 * @brief Simple test driver program for PageRank.
 */

#include <gunrock/app/ReLU/ReLU_app.cu>
#include <gunrock/app/test_base.cuh>
#include <cstdio>
#include <iostream>

using namespace gunrock;

/******************************************************************************
 * Main
 ******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct {
  /**
   * @brief the actual main function, after type switching
   * @tparam VertexT    Type of vertex identifier
   * @tparam SizeT      Type of graph size, i.e. type of edge identifier
   * @tparam ValueT     Type of edge values
   * @param  parameters Command line parameters
   * @param  v,s,val    Place holders for type deduction
   * \return hipError_t error message(s), if any
   */
  template <typename VertexT,  // Use int as the vertex identifier
            typename SizeT,    // Use int as the graph size type
            typename ValueT>   // Use int as the value type
  hipError_t
  operator()(util::Parameters &parameters, VertexT v, SizeT s, ValueT val) {
    typedef typename app::TestGraph<VertexT, SizeT, ValueT, graph::HAS_CSR>
        GraphT;
    // typedef typename GraphT::CooT CooT;

    hipError_t retval = hipSuccess;
    bool quick = parameters.Get<bool>("quick");
    bool quiet = parameters.Get<bool>("quiet");
    int len = parameters.Get<int>("len");

    double *in = new double[len], *out = new double[len];
    app::ReLU::rand_array(len, in);
    app::ReLU::ReLU(parameters, len, in, out);
    const double EPS = 1e-9;
    for (int i = 0; i < len; i++) {
//      std::cerr << "in[" << i << "]: " << in[i] << ", " << "out[" << i << "]: " << out[i] << std::endl;
      if (out[i] < -EPS) {
        std::cerr << "in[" << i << "]: " << in[i] << ", " << "out[" << i << "]: " << out[i] << std::endl;
      }
    }
    return retval;
  }
};

int main(int argc, char **argv) {
  hipError_t retval = hipSuccess;
  util::Parameters parameters("test ReLU");
  GUARD_CU(graphio::UseParameters(parameters));
  GUARD_CU(app::ReLU::UseParameters(parameters));
  GUARD_CU(app::UseParameters_test(parameters));
  GUARD_CU(parameters.Parse_CommandLine(argc, argv));
  parameters.Set("graph-type", "by-pass");
  if (parameters.Get<bool>("help")) {
    parameters.Print_Help();
    return hipSuccess;
  }
  GUARD_CU(parameters.Check_Required());

  return app::Switch_Types<app::VERTEXT_U32B |  // app::VERTEXT_U64B |
                           app::SIZET_U32B |    // app::SIZET_U64B |
                           app::VALUET_F64B |   // app::VALUET_F64B |
                           app::DIRECTED | app::UNDIRECTED>(parameters,
                                                            main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
